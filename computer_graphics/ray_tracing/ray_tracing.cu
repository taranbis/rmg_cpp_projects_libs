#include "hip/hip_runtime.h"
#include <fstream>
#include <iostream>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "ray.hpp"
#include "vec3.hpp"

#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)
void check_cuda(hipError_t result, char const* const func, const char* const file, int const line)
{
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << "at " << file << ": line = " << line
                  << ", func = "
                  << "\n";
    }
}

__global__ void render_image(vec3* fb, int max_x, int max_y)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if ((i >= max_x) || (j >= max_y)) return;

    // int pixel_index      = j * max_x * 3 + i * 3;
    // fb[pixel_index]     = float(i) / max_x;
    // fb[pixel_index + 1] = float(j) / max_y;
    // fb[pixel_index + 2] = 0.2;
}

__device__ float hit_sphere(const vec3& centre, float radius, const ray& r)
{
    vec3  oc = r.origin() - centre;
    float a = dot(r.direction(), r.direction());
    float b = 2.0f * dot(oc, r.direction());
    float c = dot(oc, oc) - radius * radius;
    float discriminant = b * b - 4.0f * a * c;
    if (discriminant < 0.0f) {
        return -1.0f;
    } else {
        return (-b - sqrt(discriminant)) / (2.0f * a);
    }
}

__device__ vec3 color(const ray& r)
{
    float t = hit_sphere(vec3(0, 0, -1), 0.5f, r);
    if (t > 0.0f) {
        vec3 N = unit_vector(r.point_at_parameter(t) - vec3(0, 0, -1));
        return 0.5f * vec3(N.x() + 1, N.y() + 1, N.z() + 1);
    }
    vec3 unit_direction = unit_vector(r.direction());
    t = 0.5f * (unit_direction.y() + 1.0f);
    return (1.0f - t) * vec3(1.0, 1.0, 1.0) + t * vec3(0.5, 0.7, 1.0);
}

__global__ void render(vec3* fb, int max_x, int max_y, vec3 lower_left_corner, vec3 horizontal, vec3 vertical,
                       vec3 origin)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) { return; }

    int   pixel_index = j * max_x + i;
    float u = float(i) / float(max_x);
    float v = float(j) / float(max_y);
    ray   r(origin, lower_left_corner + u * horizontal + v * vertical);
    fb[pixel_index] = color(r);
}

void print_image()
{
    const int nx = 200;
    const int ny = 100;
    // output as .ppm image
    std::ofstream fout("out.ppm", std::ios_base::out | std::ios_base::binary);
    fout << "P6" << std::endl << nx << ' ' << ny << std::endl << "255" << std::endl;
    for (int j = ny - 1; j >= 0; --j) {
        for (int i = 0; i < nx; ++i) {
            vec3 col(float(i) / float(nx), float(j) / float(ny), 0.2);
            int  ir = int(255.99f * col[0]);
            int  ig = int(255.99f * col[1]);
            int  ib = int(255.99f * col[2]);

            fout << ir << " " << ig << " " << ib << "\n";
        }
    }
}

int main()
{
    int    nx = 1024, ny = 512;
    int    num_pixels = nx * ny;
    size_t fb_size = num_pixels * sizeof(vec3);

    vec3* fb;
    checkCudaErrors(hipMallocManaged((void**)&fb, fb_size));

    int  tx = 8, ty = 8;
    dim3 blocks(nx / tx, ny / ty);
    dim3 threads(tx, ty);

    hipEvent_t start, stop;
    float       time;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    /**
     * \note Using “0” in this argument means that the default stream object
     * is used for synchronization.
     */
    hipEventRecord(start, 0);

    vec3 lower_left_corner(-2.0, -1.0, -1.0);
    vec3 horizontal(4.0, 0.0, 0.0);
    vec3 vertical(0.0, 2.0, 0.0);
    vec3 origin(0.0, 0.0, 0.0);

    render<<<blocks, threads>>>(fb, nx, ny, lower_left_corner, horizontal, vertical, origin);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    /**
     * \note time value is expressed as milliseconds
     *  and it is accurate to about 0.5 microseconds
     */
    hipEventElapsedTime(&time, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    checkCudaErrors(hipGetLastError());

    /**
     * \note \fn hipDeviceSynchronize lets the CPU know when the
     */
    checkCudaErrors(hipDeviceSynchronize());

    // for (int j = ny - 1; j >= 0; j--)
    // {
    //     for (int i = 0; i < nx; i++)
    //     {
    //         size_t pixel_index = j * nx + i;
    //         std::cout << "fb[" << pixel_index << "]= " << fb[pixel_index]
    //                   << " ";
    //     }
    //     std::cout << std::endl;
    // }

    // output as .ppm image
    std::ofstream fout("out.ppm", std::ios_base::out | std::ios_base::binary);
    fout << "P6" << std::endl << nx << ' ' << ny << std::endl << "255" << std::endl;
    for (int j = ny - 1; j >= 0; j--) {
        for (int i = 0; i < nx; i++) {
            size_t pixel_index = j * nx + i;
            int    ir = int(255.99 * fb[pixel_index].r());
            int    ig = int(255.99 * fb[pixel_index].g());
            int    ib = int(255.99 * fb[pixel_index].b());
            fout << (char)ir << (char)ig << (char)ib;
        }
    }
    fout.close();

    checkCudaErrors(hipFree(fb));
    std::cout << "time: " << time << std::endl;
    return 0;
}

// int main()
// {
//     vec3* fb = new vec3(3, 4, 0);
//     // fb->x(1);
//     // fb->y(2);
//     // fb->z(3);

//     fb->make_unit_vector();

//     std::cout << *fb;
//     return 0;
// }